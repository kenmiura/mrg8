/*
 * MRG8 Inner product version
 *
 *  Created on: June 29, 2017
 *      Author: Yusuke
 */

#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cmath>
#include <sys/time.h>
#include <ctime>
#include <stdint.h>

#include <mrg8_cuda.h>
#include <rng_test.h>

using namespace std;

//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
int main(int argc, char **argv)
{
    int i, N;
    double *ran, *d_ran;
    float msec, ave_msec, mrng;
    uint32_t iseed;
    hipEvent_t event[2];
    
    for (i = 0; i < 2; ++i) {
        hipEventCreate(&(event[i]));
    }

    iseed = 13579;
    if (argc > 1) {
        N = atoi(argv[1]) * 1024 * 1024;
    }
    else {
        N = 1 * 1024 * 1024;
    }

    cout << "Generating " << N << " of 64-bit floating random numbers" << endl;

    ran = new double[N];
    hipMalloc((void **)&d_ran, sizeof(double) * N);

    mrg8_cuda m(iseed);
    
    for (int TNUM = 1024; TNUM <= 1024 * 1024; TNUM *= 2) {
        ave_msec = 0;
        for (i = 0; i < ITER; ++i) {
            m.seed_init(iseed);
            hipEventRecord(event[0], 0);
            m.mrg8_inner(d_ran, N, TNUM);
            hipEventRecord(event[1], 0);
            hipDeviceSynchronize();
            hipEventElapsedTime(&msec, event[0], event[1]);
#ifdef DEBUG
            if (i == 0) {
                hipMemcpy(ran, d_ran, sizeof(double) * N, hipMemcpyDeviceToHost);
                check_rand(ran, N);
            }
#endif
            if (i > 0) {
                ave_msec += msec;
            }
        }
        ave_msec /= (ITER - 1);
        mrng = (double)(N) / ave_msec / 1000;
        cout << "MRG8_CUDA_inner with " << TNUM << " threads: " << mrng << " [million rng/sec], " << ave_msec << " [milli seconds]" << endl;
        printf("EVALUATION, MRG8_CUDA_inner, %d, , %d, %f, %f\n", TNUM, N, mrng, ave_msec);
    }
    
    hipFree(d_ran);
    delete[] ran;
    for (i = 0; i < 2; ++i) {
        hipEventDestroy(event[i]);
    }

    return 0;
}

